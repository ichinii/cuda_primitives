#include <iostream>
#include <vector>
#include <chrono>
#include <numeric>
#include "reduce.h"

int main()
{
    constexpr const std::size_t N = 1<<22;
    constexpr const std::size_t T = 128;
    // constexpr const std::size_t B = N/T;
    constexpr const std::size_t S = N * sizeof(int);

    std::cout << std::boolalpha
        << "num elements: " << N << std::endl
        << "num threads per block: " << T << std::endl
        << std::endl;

    int *data_src;
    int *data_dst;
    hipMalloc(&data_src, S);
    hipMalloc(&data_dst, S);

    auto init_data = std::vector<int>(N);
    for (std::size_t i = 0; i < N; ++i)
        init_data[i] = 1;

    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    auto test = [&] (const char *name, auto test_fn) {
        using namespace std::chrono;

        hipMemcpy(data_src, init_data.data(), S, hipMemcpyHostToDevice);

        auto start_time = steady_clock::now();
        hipEventRecord(start_event);
        test_fn();
        hipEventRecord(stop_event);
        hipEventSynchronize(stop_event);
        auto stop_time = steady_clock::now();

        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start_event, stop_event);

        auto elapsed_time2 = duration_cast<microseconds>(stop_time - start_time).count() / 1000.0f;

        int result;
        hipMemcpy(&result, data_dst, sizeof(int), hipMemcpyDeviceToHost);

        std::cout
            << "running test: '" << name << "'" << std::endl
            // << "expected: " << N << std::endl
            // << "got:      " << result << std::endl
            << "ok:       " << (N == result) << std::endl
            << "duration: " << elapsed_time << std::endl
            << "duration: " << elapsed_time2 << std::endl
            << std::endl;
    };

    auto test_cpu_0 = [&] {
        using namespace std::chrono;

        auto start_time = steady_clock::now();

        int result = 0;
        for (int i = 0; i < N; ++i)
            result += init_data[i];

        auto stop_time = steady_clock::now();
        auto elapsed_time = duration_cast<microseconds>(stop_time - start_time).count() / 1000.0f;
        std::cout
            << "running cpu test" << std::endl
            << "ok:       " << (N == result) << std::endl
            << "duration: " << elapsed_time << std::endl
            << std::endl;
    };

    auto test_cpu_1 = [&] {
        using namespace std::chrono;

        auto start_time = steady_clock::now();

        int result = std::accumulate(init_data.begin(), init_data.end(), 0);

        auto stop_time = steady_clock::now();
        auto elapsed_time = duration_cast<microseconds>(stop_time - start_time).count() / 1000.0f;
        std::cout
            << "running cpu test" << std::endl
            << "ok:       " << (N == result) << std::endl
            << "duration: " << elapsed_time << std::endl
            << std::endl;
    };

    // auto test_fn_0 = [&] {
    //     for (auto b = B; 0 < b; b/=T) {
    //         sum_0<<<b, T>>>(data_dst, data_src);
    //         std::swap(data_dst, data_src);
    //     }
    //     std::swap(data_dst, data_src);
    // };

    // auto test_fn_1 = [&] {
    //     for (auto b = B; 0 < b; b/=T) {
    //         sum_1<<<b, T, T*sizeof(int)>>>(data_dst, data_src);
    //         std::swap(data_dst, data_src);
    //     }
    //     std::swap(data_dst, data_src);
    // };

    // auto test_fn_2 = [&] {
    //     for (auto b = B; 0 < b; b/=T) {
    //         sum_2<<<b, T, T*sizeof(int)>>>(data_dst, data_src);
    //         std::swap(data_dst, data_src);
    //     }
    //     std::swap(data_dst, data_src);
    // };

    // auto test_fn_3 = [&] {
    //     for (auto b = B; 0 < b; b/=T) {
    //         sum_3<<<b, T, T*sizeof(int)>>>(data_dst, data_src);
    //         std::swap(data_dst, data_src);
    //     }
    //     std::swap(data_dst, data_src);
    // };

    // auto test_fn_4 = [&] {
    //     int t = T;
    //     for (auto b = B/2; 1 < b; b/=T*2) {
    //         sum_4<<<b, T, T*sizeof(int)>>>(data_dst, data_src);
    //         std::swap(data_dst, data_src);
    //         t /= 2;
    //     }
    //     sum_4<<<1, t/2, t*sizeof(int)>>>(data_dst, data_src);
    // };

    // auto test_fn_5 = [&] {
    //     int t = T;
    //     for (auto b = B/2; 1 < b; b/=T*2) {
    //         sum_5<<<b, T, T*sizeof(int)>>>(data_dst, data_src);
    //         std::swap(data_dst, data_src);
    //         t /= 2;
    //     }
    //     sum_3<<<1, t/2, t*sizeof(int)>>>(data_dst, data_src);
    // };

    auto test_fn_6 = [&] (std::size_t maxBlocks) {
        do_sum_6(data_dst, data_src, maxBlocks, T, N);
    };

    auto test_fn_7 = [&] (std::size_t maxBlocks) {
        do_sum_7(data_dst, data_src, maxBlocks, T, N);
    };

    auto test_reduce6 = [&] (std::size_t maxBlocks) {
        do_reduce6(data_dst, data_src, maxBlocks, T, N);
    };

    test_cpu_0();
    test_cpu_1();
    // test("sum_0", test_fn_0);
    // test("sum_1", test_fn_1);
    // test("sum_2", test_fn_2);
    // test("sum_3", test_fn_3);
    // test("sum_4", test_fn_4);
    // test("sum_5", test_fn_5);

    // for (int i = 0; i < 20; ++i) {
        int i = 12;
        int maxBlocks = 1<<i;
        std::cout << "\tthread ratio: " << (T*maxBlocks / 2816.0f) << std::endl;
        std::cout << "\tmax blocks: " << maxBlocks << std::endl << std::endl;
        // test("sum_6", [=] { test_fn_6(maxBlocks); });
        test("reduce6", [=] { test_reduce6(maxBlocks); });
        test("sum_7", [=] { test_fn_7(maxBlocks); });
    // }

    hipFree(data_src);
    hipFree(data_dst);

    return 0;
}
